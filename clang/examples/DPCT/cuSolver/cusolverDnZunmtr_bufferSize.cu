
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *tau,
          const hipDoubleComplex *c, int ldc) {
  // Start
  int buffer_size;
  hipsolverDnZunmtr_bufferSize(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/,
      uplo /*cublasFillMode_t*/, trans /*cublasOperation_t*/, m /*int*/,
      n /*int*/, a /*const cuDoubleComplex **/, lda /*int*/,
      tau /*const cuDoubleComplex **/, c /*const cuDoubleComplex **/,
      ldc /*int*/, &buffer_size /*int **/);
  // End
}
