
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const __half *alpha,
          const __half *a, int lda, long long int stridea, const __half *b,
          int ldb, long long int strideb, const __half *beta, __half *c,
          int ldc, long long int stridec, int group_count) {
  // Start
  hipblasHgemmStridedBatched(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const __half **/, a /*const __half **/, lda /*int*/,
      stridea /*long long int*/, b /*const __half **/, ldb /*int*/,
      strideb /*long long int*/, beta /*const __half **/, c /*__half **/,
      ldc /*int*/, stridec /*long long int*/, group_count /*int*/);
  // End
}
