
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, hipComplex *x, int64_t incx,
          hipComplex *y, int64_t incy, const float *c, const float *s) {
  // Start
  hipblasCsrot_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*cuComplex **/,
                 incx /*int64_t*/, y /*cuComplex **/, incy /*int64_t*/,
                 c /*const float **/, s /*const float **/);
  // End
}
