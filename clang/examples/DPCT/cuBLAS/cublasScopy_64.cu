
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const float *x, int64_t incx,
          float *y, int64_t incy) {
  // Start
  hipblasScopy_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*const float **/,
                 incx /*int64_t*/, y /*float **/, incy /*int64_t*/);
  // End
}
