// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/math/half/half_cuda11_after %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/math/half/half_cuda11_after/half_cuda11_after.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/math/half/half_cuda11_after/half_cuda11_after.dp.cpp -o %T/math/half/half_cuda11_after/half_cuda11_after.dp.o %}


#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

__global__ void kernelFuncHalfConversion() {
  half h;
  half2 h2;
  double d;
  // CHECK: h = sycl::half(d);
  h = __double2half(d);
  // CHECK: h2 = sycl::half2(h, h);
  h2 = make_half2(h, h);
}

int main() {
  kernelFuncHalfConversion<<<1, 1>>>();
  return 0;
}
